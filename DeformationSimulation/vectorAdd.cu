#include "hip/hip_runtime.h"
#include<iostream>
#include <hip/hip_runtime.h> 
#include <>

/*�˺������豸���к�����*/
__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i] + 10;
    }
}

/*��������*/
void test1(int num)
{
    /*�������������ڴ� h_A, h_B, h_C*/
    int numElements = num;
    size_t size = numElements * sizeof(float);
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    /*�����豸�ڴ� d_A��d_B��d_C */
    float* d_A = NULL;
    hipMalloc((void**)&d_A, size);
    float* d_B = NULL;
    hipMalloc((void**)&d_B, size);
    float* d_C = NULL;
    hipMalloc((void**)&d_C, size);

    /*�������ڴ����ݸ��Ƶ��豸�ڴ� h_A--d_A��h_B--d_B */
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    /*�����豸���߳����������ú˺���*/
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd << < blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    hipGetLastError();

    /*���豸�ڴ����ݸ��Ƶ������ڴ� d_C--h_C */
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    /* �ͷ��豸�ڴ� d_A d_B d_C */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    /* �����֤ */
    std::cout << "A[0]: " << (float)h_A[0] << " B[0]: " << (float)h_B[0] << " ���C[0] = A[i] + B[i] + 10: " << (float)h_C[0] << std::endl;
    std::cout << "A[1]: " << (float)h_A[1] << " B[1]: " << (float)h_B[1] << " ���C[1] = A[i] + B[i] + 10: " << (float)h_C[1] << std::endl;
    std::cout << "A[2]: " << (float)h_A[2] << " B[2]: " << (float)h_B[2] << " ���C[2] = A[i] + B[i] + 10: " << (float)h_C[2] << std::endl;

    /* �ͷ������ڴ� h_A h_B h_C */
    free(h_A);
    free(h_B);
    free(h_C);

}